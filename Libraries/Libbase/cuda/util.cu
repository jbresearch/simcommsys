#include "hip/hip_runtime.h"
/*!
 * \file
 *
 * Copyright (c) 2010 Johann A. Briffa
 *
 * This file is part of SimCommSys.
 *
 * SimCommSys is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * SimCommSys is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with SimCommSys.  If not, see <http://www.gnu.org/licenses/>.
 */

/*!
 * \file
 * \brief   CUDA utilities.
 * \author  Johann Briffa
 */

#include "cuda-all.h"
#include "sysvar.h"

namespace cuda {

//! Get the current device

int cudaGetCurrentDevice()
   {
   int device;
   cudaSafeCall(hipGetDevice(&device));
   return device;
   }

//! Get the number of multiprocessors for the given device

int cudaGetMultiprocessorCount(int device)
   {
   if (device < 0)
      device = cudaGetCurrentDevice();
   hipDeviceProp_t prop;
   cudaSafeCall(hipGetDeviceProperties(&prop, device));
   return prop.multiProcessorCount;
   }

// Get the number of cores per multiprocessor for the given device

int cudaGetMultiprocessorSize(int device)
   {
   // Structure to map SM version to # of cores per SM
   typedef struct {
      int SM; // 0xMm (hex), M = SM Major version, and m = SM minor version
      int Cores;
   } sSMtoCores;

   sSMtoCores nGpuArchCoresPerSM[] = {
         {0x10, 8}, // Tesla Generation (SM 1.0) G80 class
         {0x11, 8}, // Tesla Generation (SM 1.1) G8x class
         {0x12, 8}, // Tesla Generation (SM 1.2) G9x class
         {0x13, 8}, // Tesla Generation (SM 1.3) GT200 class
         {0x20, 32}, // Fermi Generation (SM 2.0) GF100 class
         {0x21, 48}, // Fermi Generation (SM 2.1) GF10x class
         // *** Architectures above this line not supported ***
         {0x30, 192}, // Kepler Generation (SM 3.0) GK10x class
         {0x32, 192}, // Kepler Generation (SM 3.2) GK10x class
         {0x35, 192}, // Kepler Generation (SM 3.5) GK11x class
         {0x37, 192}, // Kepler Generation (SM 3.7) GK21x class
         {0x50, 128}, // Maxwell Generation (SM 5.0) GM10x class
         {0x52, 128}, // Maxwell Generation (SM 5.2)
         {0x53, 128}, // Maxwell Generation (SM 5.3)
         {0x60,  64}, // Pascal Generation (SM 6.0)
         {0x61, 128}, // Pascal Generation (SM 6.1)
         {0x62, 128}, // Pascal Generation (SM 6.2)
         {0x70,  64}, // Volta Generation (SM 7.0)
         {0x72,  64}, // Volta Generation (SM 7.2)
         {0x75,  64}, // Volta Generation (SM 7.5)
         {-1, -1}}; // Undefined

   // If no device is specified, pick the current one
   if (device < 0)
      device = cudaGetCurrentDevice();
   // Get properties for chosen device
   hipDeviceProp_t prop;
   cudaSafeCall(hipGetDeviceProperties(&prop, device));
   // Find the SM version in the table
   for (int i = 0; nGpuArchCoresPerSM[i].SM != -1; i++)
      {
      if (nGpuArchCoresPerSM[i].SM == ((prop.major << 4) + prop.minor))
         return nGpuArchCoresPerSM[i].Cores;
      }
   std::cerr << "WARNING: SM " << prop.major << "." << prop.minor
         << " is undefined!" << std::endl;
   return -1;
   }

//! Get the amount of shared memory available per block

int cudaGetSharedMemPerBlock(int device)
   {
   if (device < 0)
      device = cudaGetCurrentDevice();
   hipDeviceProp_t prop;
   cudaSafeCall(hipGetDeviceProperties(&prop, device));
   return prop.sharedMemPerBlock;
   }

//! Get the number of registers available per block

int cudaGetRegsPerBlock(int device)
   {
   if (device < 0)
      device = cudaGetCurrentDevice();
   hipDeviceProp_t prop;
   cudaSafeCall(hipGetDeviceProperties(&prop, device));
   return prop.regsPerBlock;
   }

//! Get the maximum number of threads per block

int cudaGetMaxThreadsPerBlock(int device)
   {
   if (device < 0)
      device = cudaGetCurrentDevice();
   hipDeviceProp_t prop;
   cudaSafeCall(hipGetDeviceProperties(&prop, device));
   return prop.maxThreadsPerBlock;
   }

//! Get the warp size for the given device

int cudaGetWarpSize(int device)
   {
   if (device < 0)
      device = cudaGetCurrentDevice();
   hipDeviceProp_t prop;
   cudaSafeCall(hipGetDeviceProperties(&prop, device));
   return prop.warpSize;
   }

//! Get the clock rate in GHz for the given device

double cudaGetClockRate(int device)
   {
   if (device < 0)
      device = cudaGetCurrentDevice();
   hipDeviceProp_t prop;
   cudaSafeCall(hipGetDeviceProperties(&prop, device));
   return prop.clockRate * 1e-6f;
   }

//! Get the name for the given device

std::string cudaGetDeviceName(int device)
   {
   if (device < 0)
      device = cudaGetCurrentDevice();
   hipDeviceProp_t prop;
   cudaSafeCall(hipGetDeviceProperties(&prop, device));
   return prop.name;
   }

//! Get the amount of global memory (in bytes) for the given device

size_t cudaGetGlobalMem(int device)
   {
   if (device < 0)
      device = cudaGetCurrentDevice();
   hipDeviceProp_t prop;
   cudaSafeCall(hipGetDeviceProperties(&prop, device));
   return prop.totalGlobalMem;
   }

//! Get the compute capability for the given device

int cudaGetComputeCapability(int device)
   {
   if (device < 0)
      device = cudaGetCurrentDevice();
   hipDeviceProp_t prop;
   cudaSafeCall(hipGetDeviceProperties(&prop, device));
   return prop.major * 1000 + prop.minor;
   }

//! Get the number of CUDA-capable devices

int hipGetDeviceCount()
   {
   int devices = 0;
   cudaSafeCall(::hipGetDeviceCount(&devices));
   return devices;
   }

//! Get the version number for the device driver

int cudaGetDriverVersion()
   {
   int driverVersion = 0;
   cudaSafeCall(hipDriverGetVersion(&driverVersion));
   return driverVersion;
   }

//! Get the version number for the CUDA runtime

int cudaGetRuntimeVersion()
   {
   int runtimeVersion = 0;
   cudaSafeCall(hipRuntimeGetVersion(&runtimeVersion));
   return runtimeVersion;
   }

//! Format the given version into a printable string

std::string cudaPrettyVersion(int version)
   {
   std::ostringstream sout;
   sout << (version / 1000) << "." << (version % 100);
   return sout.str();
   }

// Returns the best GPU (with maximum GFLOPS)

int cudaGetMaxGflopsDeviceId()
   {
   int devices = hipGetDeviceCount();

   int max_gflops_device = 0;
   double max_gflops = 0;

   for (int i = 0; i < devices; i++)
      {
      double gflops = cudaGetMultiprocessorCount(i)
            * cudaGetMultiprocessorSize(i) * cudaGetClockRate(i);
      if (gflops > max_gflops)
         {
         max_gflops = gflops;
         max_gflops_device = i;
         }
      }

   return max_gflops_device;
   }

//! Kernel to determine the compute capability of current device

__global__
void getcomputemodel_kernel(value_reference<int> dev_cm)
   {
#ifndef __CUDA_ARCH__
   // should be for host code path only
   const int major = 0;
   const int minor = 0;
#else
   // device code path (any architecture model)
   const int major = __CUDA_ARCH__ / 100;
   const int minor = __CUDA_ARCH__ % 100;
#endif
   // convert into the same encoding as used for the driver/runtime version
   dev_cm() = major * 1000 + minor;
   }

//! Determine the compute capability for which this code was compiled

int cudaGetComputeModel()
   {
   // allocate space for results
   value<int> dev_cm;
   dev_cm.init();
   // call the kernel
   getcomputemodel_kernel<<<1,1>>>(dev_cm);
   // copy results back
   return dev_cm;
   }

//! Initialize the runtime and choose the best device

void cudaInitialize(std::ostream& sout)
   {
   static bool initialized = false;
   if (initialized)
      return;
   initialized = true;
   // select device to use
   int device;
   libbase::sysvar user_device("CUDA_DEVICE");
   if (user_device.is_defined())
      device = user_device.as_int();
   else
      device = cudaGetMaxGflopsDeviceId();
   cudaSafeCall(hipSetDevice(device));
   // report to user
   sout << "CUDA device: " << device << " (" << cudaGetDeviceName() << ", "
         << (cudaGetGlobalMem() >> 20) << " MiB, "
         << cudaGetMultiprocessorSize() << "×" << cudaGetMultiprocessorCount()
         << " @ " << cudaGetClockRate() << " GHz" << ", capability "
         << cudaPrettyVersion(cudaGetComputeCapability()) << ")" << std::endl;
   sout << "CUDA initialized: compute model " << cudaPrettyVersion(
         cudaGetComputeModel()) << ", cuda runtime " << cudaPrettyVersion(
         cudaGetDriverVersion()) << std::endl;
   if (cudaGetRuntimeVersion() != cudaGetDriverVersion())
      sout << "CUDA warning: this code was compiled with cuda runtime "
            << cudaPrettyVersion(cudaGetDriverVersion()) << std::endl;
   }

//! List CUDA capable devices and their properties

void cudaQueryDevices(std::ostream& sout)
   {
   // get and report the number of CUDA capable devices
   int devices = hipGetDeviceCount();
   if (devices == 0)
      {
      sout << "There is no device supporting CUDA" << std::endl;
      return;
      }
   else if (devices == 1)
      sout << "There is 1 device supporting CUDA" << std::endl;
   else
      sout << "There are " << devices << " devices supporting CUDA"
            << std::endl;

   // print driver and runtime versions
   sout << "  CUDA Driver Version:\t" << cudaPrettyVersion(
         cudaGetDriverVersion()) << std::endl;
   sout << "  CUDA Runtime Version:\t" << cudaPrettyVersion(
         cudaGetRuntimeVersion()) << std::endl;

   // print important details for all devices found
   for (int i = 0; i < devices; i++)
      {
      sout << std::endl;
      sout << "Device " << i << ": \"" << cudaGetDeviceName(i) << "\""
            << std::endl;
      sout << "  CUDA Capability:\t" << cudaPrettyVersion(
            cudaGetComputeCapability(i)) << std::endl;
      sout << "  Global memory:\t" << cudaGetGlobalMem(i) << " bytes"
            << std::endl;
      sout << "  Multiprocessors:\t" << cudaGetMultiprocessorCount(i)
            << std::endl;
      sout << "  Total Cores:\t" << cudaGetMultiprocessorSize(i)
            * cudaGetMultiprocessorCount(i) << std::endl;
      sout << "  Clock rate:\t" << cudaGetClockRate(i) << " GHz" << std::endl;

      // Get the properties for the given device
      hipDeviceProp_t prop;
      cudaSafeCall(hipGetDeviceProperties(&prop, i));
      sout << "  Memory per block:\t" << prop.sharedMemPerBlock << " bytes"
            << std::endl;
      sout << "  Threads per block:\t" << prop.maxThreadsPerBlock << std::endl;
      sout << "  Concurrent kernels:\t" << (prop.concurrentKernels ? "Yes"
            : "No") << std::endl;
      }
   }

/*! \name Get the size in bytes of statically-allocated shared memory per block
 * required by this function.
 */

size_t cudaGetSharedSize(const void* func)
   {
   hipFuncAttributes attr;
   cudaSafeCall(hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(func)));
   return attr.sharedSizeBytes;
   }

/*! \name Get the size in bytes of local memory per thread used by this
 * function.
 */

size_t cudaGetLocalSize(const void* func)
   {
   hipFuncAttributes attr;
   cudaSafeCall(hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(func)));
   return attr.localSizeBytes;
   }

//! Get the number of registers used by each thread of the given function

int cudaGetNumRegsPerThread(const void* func)
   {
   hipFuncAttributes attr;
   cudaSafeCall(hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(func)));
   return attr.numRegs;
   }

/*! \brief Get the maximum number of threads per block, beyond which a launch
 * of the function would fail.
 */

int cudaGetMaxThreadsPerBlock(const void* func)
   {
   hipFuncAttributes attr;
   cudaSafeCall(hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(func)));
   return attr.maxThreadsPerBlock;
   }

} // end namespace
