/*!
 * \file
 *
 * Copyright (c) 2010 Johann A. Briffa
 *
 * This file is part of SimCommSys.
 *
 * SimCommSys is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * SimCommSys is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with SimCommSys.  If not, see <http://www.gnu.org/licenses/>.
 */

/*!
 * \file
 * \brief   A CUDA stream
 * \author  Johann Briffa
 */

#include "cuda-all.h"

namespace cuda
{

// need to define this here as we need complete definition of event class

void
stream::wait(const event& e) const
{
    cudaSafeCall(hipStreamWaitEvent(sid, e.get_id(), 0));
}

} // namespace cuda
